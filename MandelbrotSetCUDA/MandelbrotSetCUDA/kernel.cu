#include "hip/hip_runtime.h"
// MandelbrotSet.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"
#include <math.h>
using namespace std;
const int convergenceSteps = 50;
const double divergenceLimit = 2.0;
double xMin = -(divergenceLimit + 0.5);
double xMax = (divergenceLimit + 0.5);
double yMin = -(divergenceLimit + 0.5);
double yMax = (divergenceLimit + 0.5);
double resolution = 0.01;
double zoom = 2.0;
bool isColor = true;

void magnify(int x, int y, double magnitude)
{
	if (magnitude <= 0.0)
		return;

	double size = (xMax - xMin);
	int width = glutGet(GLUT_WINDOW_WIDTH);
	int height = glutGet(GLUT_WINDOW_HEIGHT);
	double windowSize;

	if (!height)
		height = 1;
	if (!width)
		width = 1;

	if (height < width)
	{
		x -= ((width - height) / 2);
		windowSize = height;
	}
	else
	{
		y -= ((height - width) / 2);
		windowSize = width;
	}

	y = windowSize - y;
	x -= windowSize / 2.0;
	y -= windowSize / 2.0;

	double xCenter = (((double)x / windowSize) * size) + ((xMax + xMin) / 2.0);
	double yCenter = (((double)y / windowSize) * size) + ((yMax + yMin) / 2.0);
	double difference = (size * magnitude) / 2.0;

	xMin = xCenter - difference;
	xMax = xCenter + difference;
	yMin = yCenter - difference;
	yMax = yCenter + difference;
}

__global__ void addKernel(int* d_mandelbrot, int sizeI, int sizeK, point2d xBound, point2d yBound, int con, double div)
{
	int k = threadIdx.x;

	double y = yBound[0] + (yBound[1] - yBound[0]) * (double(k) / double(sizeK));
	
		for (int i = 0; i < sizeI; ++i)
		{
			double x = xBound[0] + (xBound[1] - xBound[0]) * (double(i) / double(sizeI));
			
			hipDoubleComplex z = make_hipDoubleComplex(0.0, 0.0);
			hipDoubleComplex c = make_hipDoubleComplex(x, y);
			// Sprawdzanie zbieznosci
			int j = 0;
			hipCabs(z);
			for (; j < con && hipCabs(z) < div; j++)
			{
				z = hipCadd(hipCmul(z, z), c);
			}

			/*point2d z = {0.0, 0.0};
			point2d c = {x, y};
			// Sprawdzanie zbieznosci
			int j = 0;
			
			
			for (; (j < con) && (sqrt(z[0] * z[0] + z[1] * z[1]) < div); j++)
			{
				z[0] = z[0] * z[0] - z[1] * z[1];
				z[1] = z[1] * z[0] + z[0] * z[1];
				z[0] += c[0];
				z[1] += c[1];
			}*/
			d_mandelbrot[(i * sizeI) + k] = j;
		}
}

void RenderScene(void)
{
	cout << "Entering RenderScene++" << endl;
	cout << "x=(" << xMin << ", " << xMax << ")" << endl;
	cout << "y=(" << yMin << ", " << yMax << ")" << endl;
	cout << "resolution=" << resolution << endl;
	glClear(GL_COLOR_BUFFER_BIT);
	// Czyszczenie okna aktualnym kolorem czyszcz�cym

	unsigned long long iterations = 0;
	double timePerIteration = 0.0, overallTime = 0.0;
	clock_t start = clock();
	int width, height;
	width = glutGet(GLUT_WINDOW_WIDTH);
	height = glutGet(GLUT_WINDOW_HEIGHT);

	if (width <= height)
	{
		height = width;
		//glViewport(0, (height - width) / 2, width, width);
	}
	else
	{
		width = height;
		//glViewport((width - height) / 2, 0, height, height);
	}

	int *mandelbrot = (int*)malloc((height * width) * sizeof(int));
	int *d_mandelbrot = 0;
	point2d xBound = { xMin, xMax };
	point2d yBound = { yMin, yMax };
	double* d_xBound = 0;
	double* d_yBound = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&d_mandelbrot, (height * width) * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&d_xBound, 2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&d_yBound, 2 * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
	cudaStatus = hipMemcpy(d_xBound, xBound, 2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(d_yBound, yBound, 2 * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, height >> >(d_mandelbrot, width, height, d_xBound, d_yBound, convergenceSteps, divergenceLimit);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(mandelbrot, d_mandelbrot, (height * width) * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		fprintf(stderr, hipGetErrorString(hipGetLastError()));
		goto Error;
	}
	hipFree(d_mandelbrot);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		fprintf(stderr, hipGetErrorString(hipGetLastError()));
		goto Error;
	}

	for (int i = 0; i < width; i++)
	{
		for (int k = 0; k < height; k++)
		{
			double color = (double)mandelbrot[i * height + k] / (double)convergenceSteps;
			glBegin(GL_POINTS);

			if (isColor)
			{
				int maxColor = 0xffffff;
				int clr = floor(color * (double)maxColor);
				int r, g, b;
				r = (clr & (0xff << 16)) >> 16;
				g = (clr & (0xff << 8)) >> 8;
				b = clr & (0xff);
				glColor3b(r, g, b);
			}
			else
			{
				glColor3d(color, color, color);
			}

			glVertex2i(i, k);
			glEnd();
		}
	}

	glFlush();
	free(mandelbrot);
	start = clock() - start;
	overallTime = start / (double)CLOCKS_PER_SEC;
	timePerIteration /= (double)iterations;
	// Przekazanie polece� rysuj�cych do wykonania
	cout << "Exiting RenderScene after " << overallTime << " s, " << iterations << " iterations, \ntime per iteration: "
		<< timePerIteration << " ns" << endl << endl;
	return;
Error:
	hipFree(d_mandelbrot);
}

void MyInit(void)
{
	glClearColor(1.0f, 1.0f, 1.0f, 1.0f);
	// Kolor okna wn�trza okna - ustawiono na bia�y
	cout << "\"+, -\" - zmiana rozdzielczosci rysowania\n"
		"\"z, a\" - zwiekszanie/zmniejszanie powiekszenia\n"
		"LMB, RMB - powiekszenie/pomniejszenie\n"
		"\"r, c\" - wymuszenie przerysowania/zmiany rozmiaru\n"
		"\"h\" - ten tekst pomocy\n" << endl;
}

void ChangeSize(GLsizei horizontal, GLsizei vertical)
// Parametry horizontal i vertical (szeroko�� i wysoko�� okna) s�
// przekazywane do funkcji za ka�dym razem, gdy zmieni si� rozmiar okna
{
	if (vertical == 0)
		// Zabezpieczenie pzred dzieleniem przez 0
		vertical = 1;

	glMatrixMode(GL_PROJECTION);
	// Okre�lenie uk�adu wsp�rz�dnych obserwatora

	glLoadIdentity();
	// Okre�lenie przestrzeni ograniczaj�cej

	if (horizontal <= vertical)
	{
		glViewport(0, (vertical - horizontal) / 2, horizontal, horizontal);
		glOrtho(0, horizontal, 0, horizontal, 1.0, -1.0);
	}
	else
	{
		glViewport((horizontal - vertical) / 2, 0, vertical, vertical);
		glOrtho(0, vertical, 0, vertical, 1.0, -1.0);
	}

	glMatrixMode(GL_MODELVIEW);
	// Okre�lenie uk�adu wsp�rz�dnych    

	glLoadIdentity();
}

void mouse(int button, int state, int x, int y) {
	static int leftX, leftY, rightX, rightY;
	//double xSize = xMax - xMin;
	//double ySize = yMax - yMin;
	int height = glutGet(GLUT_WINDOW_HEIGHT);
	int width = glutGet(GLUT_WINDOW_WIDTH);

	int lowerSize = height, greaterSize = width, axis = x;
	if (height > width)
	{
		lowerSize = width;
		greaterSize = height;
		axis = y;
	}

	if (button == GLUT_LEFT_BUTTON && state == GLUT_DOWN) {
		leftX = x;
		leftY = y;
		cout << "LEFT x: " << leftX << ", y: " << leftY << endl;

		if (axis < ((greaterSize - lowerSize) / 2) || axis >((greaterSize + lowerSize) / 2))
		{
			cout << "Click out of picture" << endl;
			leftX = -1;
			leftY = -1;
		}
	}
	else if (button == GLUT_RIGHT_BUTTON && state == GLUT_DOWN) {
		rightX = x;
		rightY = y;
		cout << "RIGHT x: " << rightX << ", y: " << rightY << endl;

		if (axis < ((greaterSize - lowerSize) / 2) || axis >((greaterSize + lowerSize) / 2))
		{
			cout << "Click out of picture" << endl;
			rightX = -1;
			rightY = -1;
		}
	}
	else if (state == GLUT_UP) {

		if (button == GLUT_LEFT_BUTTON && leftX >= 0 && leftY >= 0)
		{
			cout << "Magnify: " << zoom << endl;
			magnify(leftX, leftY, 1 / zoom);
			ChangeSize(glutGet(GLUT_WINDOW_WIDTH), glutGet(GLUT_WINDOW_HEIGHT));
			RenderScene();
		}
		else if (button == GLUT_RIGHT_BUTTON && rightX >= 0 && rightY >= 0)
		{
			cout << "Magnify: " << 1 / zoom << endl;
			magnify(rightX, rightY, zoom);
			ChangeSize(glutGet(GLUT_WINDOW_WIDTH), glutGet(GLUT_WINDOW_HEIGHT));
			RenderScene();
		}
	}
}

void keys(unsigned char key, int x, int y)
{
	if (key == 'r')
	{
		RenderScene();
	}
	else if (key == 'c')
	{
		isColor = !isColor;
		RenderScene();
	}
	else if (key == '+')
	{
		resolution *= 0.9;
		RenderScene();
	}
	else if (key == '-')
	{
		resolution *= 1.1;
		RenderScene();
	}
	else if (key == 'a')
	{
		if (zoom > 1.0)
		{
			zoom -= 1.0;
			cout << "Changed zoom to " << zoom << endl;
		}
		else
			zoom = 1.0;
	}
	else if (key == 'z')
	{
		if (zoom < 20.0)
		{
			zoom += 1.0;
			cout << "Changed zoom to " << zoom << endl;
		}
		else
			zoom = 20.0;
	}
	else if (key == 'h')
	{
		cout << "\n\"+, -\" - zmiana rozdzielczosci rysowania\n"
			"\"z, a\" - zwiekszanie/zmniejszanie powiekszenia\n"
			"LMB, RMB - powiekszenie/pomniejszenie\n"
			"\"r, c\" - wymuszenie przerysowania/czy kolorowac\n"
			"\"h\" - ten tekst pomocy\n" << endl;
	}
}

int main(void)
{
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);
	// Ustawienie trybu wy�wietlania
	// GLUT_SINGLE - pojedynczy bufor wy�wietlania
	// GLUT_RGBA - model kolor�w RGB

	glutCreateWindow("Mandelbrot Set CUDA");
	// Utworzenie okna i okre�lenie tre�ci napisu w nag��wku okna

	glutDisplayFunc(RenderScene);
	// Okre�lenie, �e funkcja RenderScene b�dzie funkcj� zwrotn� (callback)
	// Biblioteka GLUT b�dzie wywo�ywa�a t� funkcj� za ka�dym razem, gdy
	// trzeba b�dzie przerysowa� okno

	glutReshapeFunc(ChangeSize);
	// Dla aktualnego okna ustala funkcj� zwrotn� odpowiedzialn� za
	// zmiany rozmiaru okna

	glutKeyboardFunc(keys);
	glutMouseFunc(mouse);

	MyInit();
	// Funkcja MyInit (zdefiniowana powy�ej) wykonuje wszelkie 
	// inicjalizacje konieczneprzed przyst�pieniem do renderowania

	glutMainLoop();
	// Funkcja uruchamia szkielet biblioteki GLUT
}

/*************************************************************************************/